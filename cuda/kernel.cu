#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>

#define PATH "C:/Users/jungu/Downloads"

__global__ void make_rx_ry(float* rx, float* ry, const float* xx, const float* yy, const float theta, const int nx, const int ny, const float M_PI)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    rx[j * nx + i] = xx[i * ny + j] * cos(theta - M_PI / 2.0) + yy[i * ny + j] * sin(theta - M_PI / 2.0);
    ry[j * nx + i] = -xx[i * ny + j] * sin(theta - M_PI / 2.0) + yy[i * ny + j] * cos(theta - M_PI / 2.0);

    // change xx and yy to 1-d array first, and after this function, change rx and ry into 1-d array, threadIdx.x and blockIdx.x must be 750
}

__global__ void make_pu_ratio(float* pu, float* Ratio, const float* rx, const float* ry, const float DSD, const float DSO, const float us, const float du, const int nx, const int ny)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    pu[j * ny + i] = (((rx[j * nx] * DSD / (ry[j * ny + i] + DSO))) + us) / (-du) + 1;
    Ratio[j * ny + i] = (DSO * DSO) / ((DSO + ry[j * ny + i]) * (DSO + ry[j * ny + i]));
}

__global__ void make_pv(float* pv, const float* ry, const float* zs, const float DSD, const float DSO, const float dv, const int ny, const float vs) {

    int i = threadIdx.x;
    int j = blockIdx.x;
    int k = blockIdx.y;
    pv[k * ny * ny + i * ny + j] = ((zs[k] * DSD) / (ry[i * ny + j] + DSO) - vs) / dv + 1;

}

__global__ void interp2(float* proj, float* pu, float* pv, float* result, const int proj_idx) {
    int i = threadIdx.x;
    int j = blockIdx.x;
    int k = blockIdx.y;
    //int *x, *y;

    //x[i * 750 + j] = (0 < pu[i * 750 + j] < 1628) ? ceil(pu[i * 750 + j]) : 1;
    //y[i * 750 + j] = (0 < pv[i * 750 + j] < 1500) ? ceil(pv[i * 750 + j]) : 1;
    result[k * 750 * 750 + i * 750 + j] = proj[1500 * 1628 * proj_idx + ((int)ceil(pu[i * 750 + j]) - 1) * 1628 + (int)ceil(pv[k * 750 * 750 + i * 750 + j]) - 1];
}

__global__ void image_final(float* img, const float* Ratio, const float* result, const int nx, const int ny) {

    int j = threadIdx.x;
    int k = blockIdx.x;
    int i = blockIdx.y;
    img[i * nx * ny + nx * j + k] += Ratio[j * ny + k] * result[i * nx * ny + j * ny + k];

}



void back_projection(float* final_img, float* proj,
    int nx, int ny, int nz, float sx, float sy, float sz, int nu, int nv, float su, float sv, float DSD, float DSO,
    float off_z, float off_u, float off_v, int num_angles, float* xs, float* ys, float* zs, float* us, float* vs) {

    //clock_t start1 = clock();
    float dx = sx / nx;
    float dy = sy / ny;
    float dz = sz / nz;
    float du = 0.098;
    float dv = 0.098;
    float M_PI = 3.141592;
    int i, j;

    float* xx = (float*)malloc(nx * ny * sizeof(float));
    float* yy = (float*)malloc(nx * ny * sizeof(float));

    for (i = 0; i < nx; i++) {
        for (j = 0; j < ny; j++) {
            xx[i * ny + j] = xs[i];
            yy[i * ny + j] = ys[j];

        }
    }
    //clock_t end1 = clock();
    //printf("pre setting: %lf", (double)(end1 - start1));
    //printf("%f%f", xx[10000], yy[10000]);
    float* d_img, * d_zs, * d_xx, * d_yy, * d_proj;
    hipMalloc(&d_img, nx * ny * nz * sizeof(float));
    hipMalloc(&d_zs, nz * sizeof(float));
    hipMalloc(&d_xx, nx * ny * sizeof(float));
    hipMalloc(&d_yy, nx * ny * sizeof(float));
    hipMalloc(&d_proj, nu * nv * num_angles * sizeof(float));
    hipMemcpy(d_xx, xx, nx * ny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_yy, yy, nx * ny * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_zs, zs, nz * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_proj, proj, nu * nv * num_angles * sizeof(float), hipMemcpyHostToDevice);

    for (int proj_idx = 0; proj_idx < num_angles; proj_idx++) {

        float* d_rx, * d_ry, * d_pu, * d_ratio;
        //clock_t start2 = clock();
        hipMalloc(&d_rx, nx * ny * sizeof(float));
        hipMalloc(&d_ry, nx * ny * sizeof(float));
        hipMalloc(&d_pu, nx * ny * sizeof(float));
        hipMalloc(&d_ratio, nx * ny * sizeof(float));


        float theta = 2 * M_PI * proj_idx / num_angles;
        //clock_t end2 = clock();
        //printf("Memory copying: %lf", (double)(end2 - start2));
        //clock_t start3 = clock();
        make_rx_ry << < 750, 750 >> > (d_rx, d_ry, d_xx, d_yy, theta, nx, ny, M_PI);
        //clock_t end3 = clock();
        //float* rx = (float*)calloc(nu * nv, sizeof(float));
        //float* ry = (float*)calloc(nu * nv, sizeof(float));
        //hipMemcpy(rx, d_rx, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        //hipMemcpy(ry, d_ry, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        //printf("%f%f", rx[10000], ry[10000]);
        //printf("make rx ry: %lf", (double)(end3 - start3));
        //clock_t start4 = clock();
        make_pu_ratio << < 750, 750 >> > (d_pu, d_ratio, d_rx, d_ry, DSD, DSO, us[0], du, nx, ny);
        //clock_t end4 = clock();
        //printf("make pu ratio: %lf", (double)(end4 - start4));
        //float* pu = (float*)calloc(nx * ny, sizeof(float));
        //float* ratio = (float*)calloc(nx * ny, sizeof(float));
        //hipMemcpy(pu, d_pu, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        //hipMemcpy(ratio, d_ratio, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        //printf("%f%f", pu[11000], ratio[11000]);

        //change iz value


        float* d_pv, * d_result;

        hipMalloc(&d_pv, nx * ny * nz * sizeof(float));
        hipMalloc(&d_result, nx * ny * nz * sizeof(float));
        //clock_t start5 = clock();
        dim3 blocks1(750, 450);
        make_pv << < blocks1, 750 >> > (d_pv, d_ry, d_zs, DSD, DSO, dv, ny, vs[0]);
        //clock_t end5 = clock();
        //printf("make pv: %lf", (double)(end5 - start5));
            //float* pv = (float*)calloc(nx * ny, sizeof(float));
            //hipMemcpy(pv, d_pv, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
            //printf("%f", pv[11000]);
        //clock_t start6 = clock();
        dim3 blocks2(750, 450);
        interp2 << < blocks2, 750 >> > (d_proj, d_pu, d_pv, d_result, proj_idx);
        //float* result = (float*)calloc(nx * ny, sizeof(float));
        //hipMemcpy(result, d_result, nx * ny * sizeof(float), hipMemcpyDeviceToHost);
        //printf("%f", result[300000]);
    //clock_t end6 = clock();
    //printf("interp2: %lf", (double)(end6 - start6));
    // start7 = clock();
        dim3 blocks3(750, 450);
        image_final << < blocks3, 750 >> > (d_img, d_ratio, d_result, nx, ny);
        //float* img = (float*)calloc(nx * ny * nz, sizeof(float));
        //hipMemcpy(img, d_img, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost);
        //printf("%f", img[112800000]);
    //clock_t end7 = clock();
    //printf("image final: %lf", (double)(end7 - start7));
        hipFree(d_pv);
        hipFree(d_result);
        //printf("%f", final_img[112800000]);
        hipFree(d_rx);
        hipFree(d_ry);
        hipFree(d_pu);
        hipFree(d_ratio);
        printf("%d", proj_idx);
    }
    hipMemcpy(final_img, d_img, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost);
    //printf("%f", final_img[112800000]);
    hipFree(d_xx);
    hipFree(d_yy);
    hipFree(d_img);
    hipFree(d_zs);
}

int main(int argc, char* argv[]) {
    int num_projections = 353; // number of projections
    int index_projection = 1;     // starting point of projections

    int nx = 750; // width voxels of image
    int ny = 750; // height voxels of image
    int nz = 450; // number of images

    float sx = 150; // real width of image [mm]
    float sy = 150; // real height of image [mm]
    float sz = 90;  // real width of image [mm]

    int nu = 1628; // width of projection
    int nv = 1500; // height of projection

    float su = 147;     // real width of projection [mm]
    float sv = 159.544; // real height of projection [mm]

    float DSD = 658.45;
    float DSO = 409.70;

    float off_z = -40;
    float off_u = -41.633;
    float off_v = -74.662;
    /**array param**/

    float dx = sx / nx;
    float dy = sy / ny;
    float dz = sz / nz;
    float du = su / nu;
    float dv = sv / nv;

    int i;
    float* xs = (float*)malloc(nx * sizeof(float));
    float* ys = (float*)malloc(ny * sizeof(float));
    float* zs = (float*)malloc(nz * sizeof(float));
    float* us = (float*)malloc(nu * sizeof(float));
    float* vs = (float*)malloc(nv * sizeof(float));

    // Generate xs array
    for (i = 0; i < nx; i++) {
        xs[i] = ((-(nx - 1) / 2.0) + i) * dx;
    }

    // Generate ys array
    for (i = 0; i < ny; i++) {
        ys[i] = ((-(ny - 1) / 2.0) + i) * dy;
    }

    // Generate zs array
    for (i = 0; i < nz; i++) {
        zs[i] = ((-(nz - 1) / 2.0) + i) * dz + off_z;
    }

    // Generate us array
    for (i = 0; i < nu; i++) {
        us[i] = ((-(nu - 1) / 2.0) + i) * du + off_u;
    }

    // Generate vs array
    for (i = 0; i < nv; i++) {
        vs[i] = ((-(nv - 1) / 2.0) + i) * dv + off_v;
    }

    printf("1, finished");
    float* projection =
        (float*)calloc(num_projections * nu * nv,
            sizeof(float)); // allocate memory to projections
    float* image =
        (float*)calloc(nx * ny * nz, sizeof(float)); // allocate memory to images

    for (int i = 0; i < num_projections; i++) {
        char filename[300];
        sprintf(filename, "%s/my_input/input_%03d.raw", PATH, i + index_projection);
        FILE* fp = fopen(filename, "rb"); // load file
        if (fp == NULL) {
            fputs("File error\n", stderr);
            exit(1);
        } // check if file is loaded

        float* buffer =
            (float*)calloc(nu * nv, sizeof(float)); // allocate memory to buffer
        fread(buffer, sizeof(float), nu * nv, fp);   // read file to buffer
        fclose(fp);
        for (int j = 0; j < nu * nv; j++) {
            //            buffer[j] = buffer[j] / 28415;
            //            if (buffer[j] == 0) buffer[j] = 1;
            //            buffer[j] = -log(buffer[j]);
            projection[nu * nv * i + j] =
                buffer[j]; // copy value of buffer to projections
        }
        free(buffer);
        if (i % 10 == 0)
            printf("reading %d\n", i + index_projection);
    }
    // open input files and store to memory
    printf("done reading the file");
    //    printf("%u\n", projection[32546]);
    //    printf("Projections Loaded.\n");
    //
    back_projection(image, projection, nx, ny, nz, sx, sy, sz, nu, nv, su, sv,
        DSD, DSO, off_z, off_u, off_v, num_projections, xs, ys, zs,
        us, vs);
    printf("Backprojection Done.\n");

    for (int i = 0; i < 450; i++) {
        char filename[100];
        sprintf(filename, "%s/output/cc_%04d.raw", PATH, i);
        FILE* fp = fopen(filename, "wb"); // open file to write
        if (fp == NULL) {
            fputs("File error\n", stderr);
            exit(1);
        } // check if file is loaded
        float* buffer =
            (float*)calloc(nx * ny, sizeof(float)); // allocate memory to buffer
        for (int j = 0; j < nx * ny; j++) {
            buffer[j] = image[nx * ny * i + j]; // copy value of images to buffer
        }
        fwrite(buffer, sizeof(float), nx * ny, fp); // read file to buffer
        fclose(fp);
        free(buffer);
    } // save output images to files

    printf("Images Saved.\n");

    free(projection);
    free(image);

    return 0;
}